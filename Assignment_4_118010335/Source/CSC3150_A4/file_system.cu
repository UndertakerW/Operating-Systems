#include "hip/hip_runtime.h"
﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ __managed__ u32 gtime = 0;

// initialize the volume by setting all bytes to 0
__device__ void init_volume(FileSystem *fs) {
	for (int i = 0; i < fs->STORAGE_SIZE; i++)
		fs->volume[i] = 0;
}

// read a filename in the volume into a char*
__device__ void read_filename(FileSystem *fs, int addr, char *dest) {
	int offset = 0;
	while (fs->volume[addr + offset]) {
		dest[offset] = fs->volume[addr + offset];
		offset++;
	}
	dest[offset] = '\0';
}

// write a filename into the volume
__device__ void write_filename(FileSystem *fs, int addr, char *s) {
	int i = 0;
	while (s[i]) {
		fs->volume[addr + i] = s[i];
		i++;
	}
	fs->volume[addr + i] = '\0';
}

// read a word in the volume
__device__ uint32_t read_word(FileSystem *fs, int addr) {
	uint32_t result = 0;
	for (int i = 0; i < 4; i++)
		result += fs->volume[addr + i] << (24 - 8 * i);
	return result;
}

// write a word into the volume
__device__ void write_word(FileSystem *fs, int addr, uint32_t value) {
	for (int i = 0; i < 4; i++)
		fs->volume[addr + i] = value >> (24 - 8 * i);
}

// read a halfword in the volume
__device__ uint16_t read_halfword(FileSystem *fs, int addr) {
	uint16_t result = 0;
	for (int i = 0; i < 2; i++)
		result += fs->volume[addr + i] << (8 - 8 * i);
	return result;
}

// write a word into the volume
__device__ void write_halfword(FileSystem *fs, int addr, short value) {
	for (int i = 0; i < 2; i++)
		fs->volume[addr + i] = value >> (8 - 8 * i);
}

// update the bitmap
__device__ void update_bitmap(FileSystem *fs) {
	// update the superblock (bit map)
	int filled_bytes_num = fs->block_count / 8;
	// filled bytes = 0b 1111 1111
	for (int i = 0; i < filled_bytes_num; i++)
		fs->volume[i] = 0b11111111;
	// half-filled byte = 0b ???? ???? (could be 0)
	int half_filled_byte = 0;
	for (int i = 0; i < fs->block_count % 8; i++)
		half_filled_byte += 1 << (7 - i);
	fs->volume[filled_bytes_num] = half_filled_byte;
	// unfilled bytes = 0b 0000 0000
	for (int i = filled_bytes_num + 1; i < fs->SUPERBLOCK_SIZE; i++)
		fs->volume[i] = 0;
}

// compact the volume
__device__ int compact(FileSystem *fs, int frag_start, int frag_size) {
	int frag_end = frag_start + frag_size - 1;
	int move_start = (frag_end + 1) * fs->STORAGE_BLOCK_SIZE + fs->FILE_BASE_ADDRESS;
	int move_size = (fs->block_count - 1 - frag_end) * fs->STORAGE_BLOCK_SIZE;
	// move the subsequent data to fill up the fragment
	for (int i = 0; i < move_size; i++) {
		int from = move_start + i;
		int to = frag_start * fs->STORAGE_BLOCK_SIZE + fs->FILE_BASE_ADDRESS + i;
		fs->volume[to] = fs->volume[from];
	}
	// update the FCBs
	for (int i = 0; i <= fs->file_count; i++) {
		int fcb_addr = read_FCB_address(fs, i);
		if (fcb_addr > frag_start) {
			int fcb_addr_new = fcb_addr - frag_size;
			write_FCB_address(fs, i, fcb_addr_new);
		}
	}
	// update the block count
	fs->block_count -= frag_size;
	// update the bit map
	update_bitmap(fs);
}

// get the length of a string
__device__ int get_length(const char* ptr) {
	int length = 0;
	while (*ptr++)
		length++;
	length++;
	return length;
}

// search the FCBs for a given filename, return its address if found
__device__ int find_filename(FileSystem *fs, const char *filename) {
	int filename_length = get_length(filename);
	if (filename_length > 20) {
		printf("Error: filename \"%s\" is over %d characters\n", filename, fs->MAX_FILENAME_SIZE);
	}
	// search among the FCBs
	char *fcb_filename = (char *)malloc(20 * sizeof(char));
	for (int i = 0; i < fs->file_count; i++) {
		int base_addr = fs->SUPERBLOCK_SIZE + i * fs->FCB_SIZE;
		read_filename(fs, base_addr, fcb_filename);
		bool found = true;
		for (int i = 0; i < filename_length; i++) {
			if (fcb_filename[i] != filename[i]) {
				found = false;
				break;
			}
		}
		if (found)
			return i;
	}
	free(fcb_filename);
	// if not found
	return -1;
}

// update the FCB filename
__device__ void write_FCB_filename(FileSystem *fs, int fcb_num, char* filename) {
	int base_addr = fs->SUPERBLOCK_SIZE + fcb_num * fs->FCB_SIZE;
	write_filename(fs, base_addr, filename);
}

// update the FCB address
__device__ void write_FCB_address(FileSystem *fs, int fcb_num, uint16_t address) {
	write_halfword(fs, fs->SUPERBLOCK_SIZE + fcb_num * fs->FCB_SIZE + fs->MAX_FILENAME_SIZE, address);
}

// update the FCB size
__device__ void write_FCB_size(FileSystem *fs, int fcb_num, uint16_t size) {
	write_halfword(fs, fs->SUPERBLOCK_SIZE + fcb_num * fs->FCB_SIZE + fs->MAX_FILENAME_SIZE + sizeof(uint16_t), size);
}

// update the FCB modified time
__device__ void write_FCB_mod_time(FileSystem *fs, int fcb_num, uint32_t time) {
	write_word(fs, fs->SUPERBLOCK_SIZE + fcb_num * fs->FCB_SIZE + fs->MAX_FILENAME_SIZE + sizeof(uint32_t), time);
}

// read the FCB filename into a char*
__device__ void read_FCB_filename(FileSystem *fs, int fcb_num, char* dest) {
	int base_addr = fs->SUPERBLOCK_SIZE + fcb_num * fs->FCB_SIZE;
	read_filename(fs, base_addr, dest);
}

// read the FCB address
__device__ uint16_t read_FCB_address(FileSystem *fs, int fcb_num) {
	return read_halfword(fs, fs->SUPERBLOCK_SIZE + fcb_num * fs->FCB_SIZE + fs->MAX_FILENAME_SIZE);
}

// read the FCB size
__device__ uint16_t read_FCB_size(FileSystem *fs, int fcb_num) {
	return read_halfword(fs, fs->SUPERBLOCK_SIZE + fcb_num * fs->FCB_SIZE + fs->MAX_FILENAME_SIZE + sizeof(uint16_t));
}

// read the FCB modified time
__device__ uint32_t read_FCB_mod_time(FileSystem *fs, int fcb_num) {
	return read_word(fs, fs->SUPERBLOCK_SIZE + fcb_num * fs->FCB_SIZE + fs->MAX_FILENAME_SIZE + sizeof(uint32_t));
}

// initialize the file system
__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
	int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
	int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE,
	int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS) {
	// init variables
	fs->volume = volume;
	fs->file_count = 0;
	fs->block_count = 0;

	// init constants
	fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
	fs->FCB_SIZE = FCB_SIZE;
	fs->FCB_ENTRIES = FCB_ENTRIES;
	fs->STORAGE_SIZE = VOLUME_SIZE;
	fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
	fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
	fs->MAX_FILE_NUM = MAX_FILE_NUM;
	fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
	fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;
	fs->MAX_BLOCK_NUM = MAX_FILE_SIZE / STORAGE_BLOCK_SIZE;

	// init volume
	init_volume(fs);
}

// open a file and return a file pointer
__device__ u32 fs_open(FileSystem *fs, char *s, int op) {
	// search the FCBs for the filename
	int fcb_num = find_filename(fs, s);
	// if the filename exists
	if (fcb_num != -1) {
		int file_addr = read_FCB_address(fs, fcb_num);
		return file_addr;
	}
	// else if the filename does not exist
	else if (fcb_num == -1) {
		// if operation is write, create a new file
		if (op == G_WRITE) {
			if (fs->file_count >= fs->MAX_FILE_NUM) {
				printf("Error: the number of files reaches %d\n", fs->MAX_FILE_NUM);
				return 0x80000000;
			}
			// update the FCB
			write_FCB_filename(fs, fs->file_count, s);
			write_FCB_address(fs, fs->file_count, fs->block_count);
			write_FCB_size(fs, fs->file_count, 0);
			write_FCB_mod_time(fs, fs->file_count, gtime);
			// increase gtime by 1
			gtime++;
			// increase file count by 1
			fs->file_count++;
			// return a pointer to the next free block
			return fs->block_count;
		}
		// if operation is read
		else if (op == G_READ) {
			printf("Error: file \"%s\" does not exist.\n", s);
			return 0x80000000;
		}
	}
}

// read the content of a file into the result buffer
__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp) {
	// if fp is over the boundary
	if (fp > fs->MAX_BLOCK_NUM)
		printf("Error: file address \"%x\" is over the boundary.\n", fp);
	// else if (fp + blocks to read) is over the boundary
	else if (fp + (size - 1) / fs->STORAGE_BLOCK_SIZE + 1 > fs->MAX_BLOCK_NUM)
		printf("Error: size to read \"%d\" is over the boundary.\n", size);
	// else if fp is in the boundary, read the data into the result buffer
	else {
		for (int i = 0; i < size; i++)
			output[i] = fs->volume[fs->FILE_BASE_ADDRESS + fp * fs->STORAGE_BLOCK_SIZE + i];
	}
}

// write the content of the input buffer into a file
__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp) {
	int blocks_occupied = (size - 1) / fs->STORAGE_BLOCK_SIZE + 1;
	// invalid case 1: if fp is over the boundary
	if (fp > fs->MAX_BLOCK_NUM) {
		printf("Error: file address \"%x\" is over the boundary.\n", fp);
		return -1;
	}
	// invalid case 2: if fp points to some free block other than the first one
	if (fp > fs->block_count) {
		printf("Error: file address \"%x\" is empty.\n", fp);
		return -1;
	}
	// invalid case 3: if size > the max size of a file
	if (size > fs->MAX_FILE_SIZE / fs->MAX_FILE_NUM) {
		printf("Error: file size \"%d\" is over the limit.\n", size);
		return -1;
	}
	// find the corresponding FCB
	int fcb_num = -1;
	for (int i = 0; i < fs->file_count; i++) {
		u32 file_addr = read_FCB_address(fs, i);
		if (file_addr == fp) {
			fcb_num = i;
			break;
		}
	}
	// invalid case 4: FCB does not exist
	if (fcb_num == -1) {
		printf("Error: FCB for fp \"%x\" does not exist.\n", fp);
		return -1;
	}
	// valid case 1: if fp points to the first free block, then this is an empty file
	if (fp == fs->block_count) {
		// invalid case 5: insufficient storage
		if (blocks_occupied > fs->MAX_BLOCK_NUM - fs->block_count) {
			printf("Error: insufficient storage.\n");
			return -1;
		}
		// directly write the data into the volume
		for (int i = 0; i < size; i++)
			fs->volume[fs->FILE_BASE_ADDRESS + fs->block_count * fs->STORAGE_BLOCK_SIZE + i] = input[i];
		// update the block count
		fs->block_count += blocks_occupied;
	}
	// if fp points to a occupied block, then this is a non-empty file
	else if (fp < fs->block_count) {
		int file_size = read_FCB_size(fs, fcb_num);
		int file_blocks_occupied = (file_size - 1) / fs->STORAGE_BLOCK_SIZE + 1;
		// valid case 2: if the blocks of the original file are enough
		if (blocks_occupied <= file_blocks_occupied) {
			// write the data into the volume
			for (int i = 0; i < size; i++)
				fs->volume[fs->FILE_BASE_ADDRESS + fp * fs->STORAGE_BLOCK_SIZE + i] = input[i];
			// detect and eliminate fragment
			int frag_size = file_blocks_occupied - blocks_occupied;
			if (frag_size > 0) {
				int frag_start = fp + blocks_occupied;
				compact(fs, frag_start, frag_size);
			}
		}
		// valid case 3: if the blocks of the original file are not enough
		else {
			// invalid case 5: insufficient storage
			if (blocks_occupied - file_blocks_occupied > fs->MAX_BLOCK_NUM - fs->block_count) {
				printf("Error: insufficient storage.\n");
				return -1;
			}
			// compact the volume
			compact(fs, fp, file_blocks_occupied);
			// update the FCB address
			write_FCB_address(fs, fcb_num, fs->block_count);
			// write the data into the volume
			for (int i = 0; i < size; i++)
				fs->volume[fs->FILE_BASE_ADDRESS + fs->block_count * fs->STORAGE_BLOCK_SIZE + i] = input[i];
			// update the block count
			fs->block_count += blocks_occupied;
		}
	}
	// update the FCB
	write_FCB_size(fs, fcb_num, size);
	write_FCB_mod_time(fs, fcb_num, gtime);
	// increase gtime by 1
	gtime++;
	// update the bit map
	update_bitmap(fs);
}

// list out the files
__device__ void fs_gsys(FileSystem *fs, int op) {
	bool *printed = (bool *)malloc(fs->file_count * sizeof(bool));
	char *filename = (char *)malloc(fs->MAX_FILENAME_SIZE * sizeof(char));
	for (int i = 0; i < fs->file_count; i++)
		printed[i] = false;
	// sort by modified time
	if (op == LS_D) {
		printf("===sort by modified time===\n");
		int index;
		int current_mod_time;
		int max_mod_time;
		for (int i = 0; i < fs->file_count; i++) {
			max_mod_time = -1;
			for (int j = 0; j < fs->file_count; j++) {
				if (!printed[j]) {
					current_mod_time = read_FCB_mod_time(fs, j);
					if (current_mod_time > max_mod_time) {
						max_mod_time = current_mod_time;
						index = j;
					}
				}
			}
			read_FCB_filename(fs, index, filename);
			printf("%s\n", filename);
			printed[index] = true;
		}
	}
	// sort by file size
	else if (op == LS_S) {
		printf("===sort by file size===\n");
		int index;
		int current_size;
		int max_size;
		for (int i = 0; i < fs->file_count; i++) {
			max_size = -1;
			for (int j = 0; j < fs->file_count; j++) {
				if (!printed[j]) {
					current_size = read_FCB_size(fs, j);
					if (current_size > max_size) {
						max_size = current_size;
						index = j;
					}
				}
			}
			read_FCB_filename(fs, index, filename);
			printf("%s %d\n", filename, read_FCB_size(fs, index));
			printed[index] = true;
		}
	}
	free(printed);
	free(filename);
}

// remove a file
__device__ void fs_gsys(FileSystem *fs, int op, char *s) {
	if (op == RM) {
		int fcb_num = find_filename(fs, s);
		//if the file exists
		if (fcb_num != -1) {
			// get the file size and number of blocks occupied
			int file_size = read_FCB_size(fs, fcb_num);
			int block_occupied = (file_size - 1) / fs->STORAGE_BLOCK_SIZE + 1;
			// get the file address
			int file_address = read_FCB_address(fs, fcb_num);
			// compact the volume
			compact(fs, file_address, block_occupied);
			// compact the FCBs
			for (int i = fcb_num; i < fs->file_count; i++) {
				for (int j = 0; j < fs->FCB_SIZE; j++)
					fs->volume[fs->SUPERBLOCK_SIZE + i * fs->FCB_SIZE + j] = fs->volume[fs->SUPERBLOCK_SIZE + (i + 1) * fs->FCB_SIZE + j];
			}
			// update the file count
			fs->file_count--;
			// update the bit map
			update_bitmap(fs);
		}
		//if the file does not exist
		else if (fcb_num == -1)
			printf("Error: file \"%s\" does not exist.\n", s);
	}
}